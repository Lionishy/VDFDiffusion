#include "hip/hip_runtime.h"

#include "ThomsonSolver.cuh"

#include <hip/hip_runtime.h>


template <typename T>
__global__ void set_test_matrix(T *mem_dev, size_t size) {
	T *a = mem_dev, *b = mem_dev + size, *c = mem_dev + 2 * size, *d = mem_dev + 3 * size, *x = mem_dev + 4*size;
	a[0] = 0.f; b[0] = 3.f; c[0] = 1.f; d[0] = 4.f;
	for (size_t idx = 1; idx != size-1; ++idx) {
		a[idx] = 1.f; b[idx] = 2.f; c[idx] = 1.f; d[idx] = 4.f;
	}
	a[size - 1] = 1.f; b[size - 1] = 3.f; c[size - 1] = 0.f; d[size - 1] = 4.f;

	for (size_t idx = 0; idx != size; ++idx)
		x[idx] = -1.f;
}

template <typename T>
__global__ void thomson_sweep_test_kernell(T *mem_dev, size_t size) {
	iki::math::device::thomson_sweep(mem_dev, mem_dev + size, mem_dev + 2 * size, mem_dev + 3 * size, mem_dev + 4 * size, size);
}

#include <iostream>
#include <vector>
#include <algorithm>

int main() {
	using namespace std;

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		cout << "Error in starting cuda device" << endl;
		goto End;
	}

	//cuda matrix data
	size_t size = 1000;
	double *mem_dev = NULL;// 5 * size =>  *a_dev, *b_dev, *c_dev, *d_dev, *x_dev;
	if (hipSuccess != hipMalloc((void **)&mem_dev, 5 * size * sizeof(double))) {
		cout << "Can't allocate enought device memory!" << endl;
		goto Clear;
	}

	set_test_matrix<<<1,1>>>(mem_dev, size);
	hipDeviceSynchronize();

	cudaStatus = hipGetLastError();
	if (hipSuccess != cudaStatus) {
		cout << "Kernell launch failed: " << hipGetErrorString(cudaStatus) << endl;
		goto Clear;
	}

	thomson_sweep_test_kernell<<<1,1>>>(mem_dev, size);
	hipDeviceSynchronize();
	{
		vector<double> result(size);
		if (hipSuccess != hipMemcpy(result.data(), mem_dev + 4 * size, size * sizeof(double), hipMemcpyDeviceToHost)) {
			cout << "Error while memory copy from device to host!" << endl;
			goto Clear;
		}

		for_each(begin(result), end(result), [] (auto x) { std::cout << x << '\n'; });
	}

Clear:;
	if (NULL != mem_dev) hipFree(mem_dev);
	if (hipSuccess != hipDeviceReset()) {
		cout << "Error in device process termination!" << endl;
	}


End:;
	return 0;
}