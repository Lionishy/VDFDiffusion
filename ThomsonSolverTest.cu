#include "hip/hip_runtime.h"

#include "ThomsonSolver.cuh"

#include <hip/hip_runtime.h>

template <typename T>
__device__ void calculate_tridiagonal_matrix(T const *f_dev, T const *d_dev, T *tsa_dev, T *tsb_dev, T *tsc_dev, T *tsd_dev, size_t size, T r) {
	tsa_dev[0] = T(0); 
	tsb_dev[0] = d_dev[0] * r / 2 + T(1); 
	tsc_dev[0] = -d_dev[0] * r / 2;
	tsd_dev[0] = f_dev[0] + r / 2 * d_dev[0] * (f_dev[1] - f_dev[0]);

	for (size_t idx = 1; idx != size-2; ++idx) {
		tsa_dev[idx] = -r / 2 * d_dev[idx - 1];
		tsb_dev[idx] = r / 2 * (d_dev[idx - 1] + d_dev[idx]) + T(1);
		tsc_dev[idx] = -r / 2 * d_dev[idx];
		tsd_dev[idx] = f_dev[idx] + r / 2 * (f_dev[idx-1]*d_dev[idx-1] - f_dev[idx]*(d_dev[idx-1]+d_dev[idx]) + f_dev[idx+1]*d_dev[idx]);
	}

	tsa_dev[size - 2] = -r / 2 * d_dev[size - 3];
	tsb_dev[size - 2] = r / 2 * (d_dev[size - 3] + d_dev[size-2]) + T(1);
	tsc_dev[size - 2] = T(0);
	tsd_dev[size - 2] = f_dev[size - 2] + r / 2 * (f_dev[size - 3] * d_dev[size - 3] - f_dev[size - 2] * (d_dev[size - 3] + d_dev[size - 2]) + 2 * f_dev[size - 1] * d_dev[size - 2]);
}

template <typename T>
__device__ void set_initial_state(T *f_dev, T *d_dev, size_t size) {
	T grad = T(1) / (size-1);
	f_dev[0] = T(1); f_dev[size - 1] = T(0);
	d_dev[0] = d_dev[size - 1] = T(1);
	for (size_t idx = 1; idx != size - 1; ++idx) {
		f_dev[idx] = T(1) - grad * idx;
		d_dev[idx] = T(1);
	}
}

template <typename T>
__global__ void thomson_sweep_test_kernell(T *f_dev, T *d_dev, T *tsa_dev, T *tsb_dev, T *tsc_dev, T *tsd_dev, T *tsx_dev, size_t size, size_t loop_count) {
	set_initial_state(f_dev, d_dev, size);
	for (; loop_count != 0; --loop_count) {
		calculate_tridiagonal_matrix(f_dev, d_dev, tsa_dev, tsb_dev, tsc_dev, tsd_dev, size, T(1.));
		iki::math::device::thomson_sweep(tsa_dev, tsb_dev, tsc_dev, tsd_dev, tsx_dev, size-1);
		for (size_t idx = 0; idx != size - 1; ++idx)
			f_dev[idx] = tsx_dev[idx];
	}
}

#include <iostream>
#include <vector>
#include <algorithm>
#include <vector>
#include <fstream>

int main() {
	using namespace std;

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		cout << "Error in starting cuda device" << endl;
		goto End;
	}

	{
		size_t size = 1024;
		vector<float> f_next(size);

		//cuda function data
		float *f_dev = NULL, *d_dev = NULL;
		//cuda thomson sweep method data
		float *tsa_dev = NULL, *tsb_dev = NULL, *tsc_dev = NULL, *tsd_dev = NULL, *tsx_dev = NULL;

		if (hipSuccess != hipMalloc((void **)&f_dev, size * sizeof(float))) {
			cout << "Can't allocate memory for function: " << size * sizeof(float) / 1024 << " Kb" << endl;
			goto Clear;
		}
		if (hipSuccess != hipMalloc((void **)&d_dev, size * sizeof(float))) {
			cout << "Can't allocate memory for diffusion coefficients: " << size * sizeof(float) / 1024 << " Kb" << endl;
			goto Clear;
		}
		if (
			hipSuccess != hipMalloc((void **)&tsa_dev, size * sizeof(float))
			|| hipSuccess != hipMalloc((void **)&tsb_dev, size * sizeof(float))
			|| hipSuccess != hipMalloc((void **)&tsc_dev, size * sizeof(float))
			|| hipSuccess != hipMalloc((void **)&tsd_dev, size * sizeof(float))
			|| hipSuccess != hipMalloc((void **)&tsx_dev, size * sizeof(float))
		) {
			cout << "Can't allocate memory for thomson sweep algorithm: " << 5 * size * sizeof(float) / 1024 << " Kb" << endl;
			goto Clear;
		}

		thomson_sweep_test_kernell<<<1,1>>>(f_dev,d_dev,tsa_dev,tsb_dev,tsc_dev,tsd_dev,tsx_dev,size,200000u);
		if (hipSuccess != hipGetLastError()) {
			cout << "Kernell launch failed: " << hipGetErrorString(cudaStatus) << endl;
			goto Clear;
		}
		else {
			cout << "Calculation Success!" << endl;
			if (hipSuccess != hipMemcpy(f_next.data(), f_dev, size * sizeof(float), hipMemcpyDeviceToHost)) {
				cout << "Memory copy device->host failed!" << endl;
			}
			else {
				ofstream ascii_out("./data/f.txt");
				for (auto f : f_next) {
					ascii_out << f << '\n';
				}
			}
		}


	Clear:;
		if (f_dev != NULL) hipFree(f_dev);
		if (d_dev != NULL) hipFree(d_dev);
		if (tsa_dev != NULL) hipFree(tsa_dev);
		if (tsb_dev != NULL) hipFree(tsb_dev);
		if (tsc_dev != NULL) hipFree(tsc_dev);
		if (tsd_dev != NULL) hipFree(tsd_dev);
		if (tsx_dev != NULL) hipFree(tsx_dev);
	}

	

	/*size_t size = 1000;
	double *mem_dev = NULL;// 5 * size =>  *a_dev, *b_dev, *c_dev, *d_dev, *x_dev;
	if (hipSuccess != hipMalloc((void **)&mem_dev, 5 * size * sizeof(double))) {
		cout << "Can't allocate enought device memory!" << endl;
		goto Clear;
	}

	set_test_matrix<<<1,1>>>(mem_dev, size);
	hipDeviceSynchronize();

	cudaStatus = hipGetLastError();
	if (hipSuccess != cudaStatus) {
		cout << "Kernell launch failed: " << hipGetErrorString(cudaStatus) << endl;
		goto Clear;
	}

	thomson_sweep_test_kernell<<<1,1>>>(mem_dev, size);
	hipDeviceSynchronize();
	{
		vector<double> result(size);
		if (hipSuccess != hipMemcpy(result.data(), mem_dev + 4 * size, size * sizeof(double), hipMemcpyDeviceToHost)) {
			cout << "Error while memory copy from device to host!" << endl;
			goto Clear;
		}

		for_each(begin(result), end(result), [] (auto x) { std::cout << x << '\n'; });
	}

Clear:;
	if (NULL != mem_dev) hipFree(mem_dev);
	if (hipSuccess != hipDeviceReset()) {
		cout << "Error in device process termination!" << endl;
	}*/


	if (hipSuccess != hipDeviceReset()) {
		cout << "Error in device process termination!" << endl;
	}
End:;
	return 0;
}