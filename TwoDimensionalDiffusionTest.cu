#include "hip/hip_runtime.h"
#include "ForwardStepKernel.cuh"
#include "CorrectionStepKernel.cuh"
#include "ThomsonSolverKernel.cuh"
#include "Transpose.cuh"

#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <chrono>
#include <utility>
#include <cmath>

template <typename T>
void initial_sin_wave(std::vector<T> &f, size_t x_size, size_t y_size, int N) {
	auto const PI = T(3.14159265358979323);
	for (size_t y_idx = 0; y_idx != y_size; ++y_idx)
		for (size_t x_idx = 0; x_idx != x_size; ++x_idx)
			f[x_idx * y_size + y_idx] = std::sin((PI * N) / (x_size-1)*x_idx);
}

template <typename T>
void initial_y_slope(std::vector<T> &f, size_t x_size, size_t y_size) {
	T grad = T(1) / (y_size - 1);
	for (size_t y_idx = 0; y_idx != y_size; ++y_idx)
		for (size_t x_idx = 0; x_idx != x_size; ++x_idx)
			f[x_idx * y_size + y_idx] = T(1) - grad * y_idx;
}

template <typename T>
void initial_x_dfc(std::vector<T> &dfc, size_t x_size, size_t y_size) {
	for (size_t y_idx = 0; y_idx != y_size; ++y_idx)
		for (size_t x_idx = 1; x_idx != x_size-2; ++x_idx)
			dfc[y_idx + x_idx * y_size] = T(1);
}

template <typename T>
void initial_y_dfc(std::vector<T> &dfc, size_t x_size, size_t y_size) {
	for (size_t x_idx = 1; x_idx != x_size; ++x_idx)
		for (size_t y_idx = 0; y_idx != y_size; ++y_idx)
			dfc[y_idx + x_idx * y_size] = T(1);
}

template <unsigned tile_dim, unsigned block_rows, typename T>
hipError_t cycle_transpose(T **f_prev, T **f_curr, T **f_tmp, size_t x_size, size_t y_size, size_t &matrix_shift) {
	hipError_t cudaStatus;

	float *f_prev_full = *f_prev - matrix_shift, *f_curr_full = *f_curr - matrix_shift, *f_tmp_full = *f_tmp - matrix_shift;
	dim3 grid(x_size / tile_dim, y_size / tile_dim), threads(tile_dim, block_rows);
	iki::math::device::transpose_kernell<tile_dim, block_rows><<<grid, threads>>>(f_tmp_full, f_curr_full, x_size, y_size);
	if (hipSuccess != (cudaStatus = hipGetLastError()))
		return cudaStatus;

	iki::math::device::transpose_kernell<tile_dim, block_rows><<<grid, threads>>>(f_curr_full, f_prev_full, x_size, y_size);
	if (hipSuccess != (cudaStatus = hipGetLastError()))
		return cudaStatus;

	auto rotate_tmp = f_prev_full;
	f_prev_full = f_curr_full;
	f_curr_full = f_tmp_full;
	f_tmp_full = rotate_tmp;

	matrix_shift = x_size + 1;
	*f_prev = f_prev_full + matrix_shift;
	*f_curr = f_curr_full + matrix_shift;
	*f_tmp = f_tmp_full + matrix_shift;

	return cudaStatus;
}

template <typename T>
hipError_t iteration_step(T **f_prev, T **f_curr, T **f_tmp, T *x_dfc, T *y_dfc, T *a, T *b, T *c, T *d, T rx, T ry, size_t x_size, size_t y_size) {
	hipError_t cudaStatus;
	int threads, blockDim;
	size_t matrix_shift = x_size + 1;

	blockDim = 1; threads = x_size - 2;
	iki::diffusion::device::forward_step_multisolver_kernel<<<blockDim, threads>>>(*f_prev, x_dfc, y_dfc, a, b, c, d, rx, ry, x_size - 2, y_size, x_size);
	if (hipSuccess != (cudaStatus = hipGetLastError()))
		return cudaStatus;

	iki::math::device::thomson_multisolver_kernell<<<blockDim, threads>>>(a, b, c, d, *f_curr, x_size - 2, y_size);
	if (hipSuccess != (cudaStatus = hipGetLastError()))
		return cudaStatus;

	if (hipSuccess != (cudaStatus = cycle_transpose<32u, 8u>(f_prev, f_curr, f_tmp, x_size, y_size, matrix_shift)))
		return cudaStatus;

	blockDim = 1; threads = y_size - 2;
	iki::diffusion::device::correction_step_multisolver_kernel<<<blockDim, threads>>>(*f_prev, *f_curr, y_dfc, a, b, c, d, ry, y_size - 2, x_size);
	if (hipSuccess != (cudaStatus = hipGetLastError()))
		return cudaStatus;

	iki::math::device::thomson_multisolver_kernell<<<blockDim, threads>>>(a, b, c, d, *f_curr, y_size - 2, x_size);
	if (hipSuccess != (cudaStatus = hipGetLastError()))
		return cudaStatus;

	if (hipSuccess != (cudaStatus = cycle_transpose<32u, 8u>(f_prev, f_curr, f_tmp, y_size, x_size, matrix_shift)))
		return cudaStatus;

	std::swap(*f_prev, *f_curr);
	return cudaStatus;
}

int main() {
	using namespace std;
	using namespace iki;

	hipError_t cudaStatus;
	float *gm_dev = NULL;
	size_t x_size = 1024, y_size = 1024;
	vector<float> f(x_size * y_size), x_diffusion(x_size * y_size), y_diffusion(x_size * y_size);
	initial_y_slope(f, x_size, y_size); initial_x_dfc(x_diffusion, x_size, y_size); initial_y_dfc(y_diffusion, y_size, x_size);
	float rx = 10.0f, ry = 10.0f;

	if (hipSuccess != (cudaStatus = hipSetDevice(0))) {
		cerr << "Error in starting cuda device: " << endl;
		cerr << cudaStatus << " -- " << hipGetErrorString(cudaStatus) << endl;
		goto End;
	}

	if (hipSuccess != (cudaStatus = hipMalloc((void **)&gm_dev, 9 * x_size * y_size * sizeof(float)))) {
		cerr << "Can't allocate global device memory of " << (9*x_size*y_size*sizeof(float)/1024) << " Kb: " << endl;
		cerr << cudaStatus << " -- " << hipGetErrorString(cudaStatus) << endl;
		goto Clear;
	}
	else {
		cerr << (9 * x_size * y_size * sizeof(float) / 1024) << " Kb: " << " successfully allocated!" << endl;
	}

	if (hipSuccess != (cudaStatus = hipMemcpy(gm_dev, f.data(), x_size * y_size * sizeof(float), hipMemcpyHostToDevice))) {
		cerr << "Can't copy data from f to device:" << endl;
		cerr << cudaStatus << " -- " << hipGetErrorString(cudaStatus) << endl;
		goto Clear;
	}

	if (hipSuccess != (cudaStatus = hipMemcpy(gm_dev + x_size * y_size, f.data(), x_size * y_size * sizeof(float), hipMemcpyHostToDevice))) {
		cerr << "Can't copy data from f to device:" << endl;
		cerr << cudaStatus << " -- " << hipGetErrorString(cudaStatus) << endl;
		goto Clear;
	}

	if (hipSuccess != (cudaStatus = hipMemcpy(gm_dev + 3 * x_size * y_size, x_diffusion.data(), x_size * y_size * sizeof(float), hipMemcpyHostToDevice))) {
		cout << "Can't copy data from x_dfc to device:" << endl;
		cout << cudaStatus << " -- " << hipGetErrorString(cudaStatus) << endl;
		goto Clear;
	}

	if (hipSuccess != (cudaStatus = hipMemcpy(gm_dev + 4 * x_size * y_size, y_diffusion.data(), x_size * y_size * sizeof(float), hipMemcpyHostToDevice))) {
		cerr << "Can't copy data from y_dfc to device:" << endl;
		cerr << cudaStatus << " -- " << hipGetErrorString(cudaStatus) << endl;
		goto Clear;
	}

	{
		size_t matrix_size = x_size * y_size, matrix_shift = y_size + 1;
		float *f_prev = gm_dev + y_size + 1, *f_curr = f_prev + matrix_size, *f_tmp = f_curr + matrix_size, *x_dfc = f_tmp + matrix_size, *y_dfc = x_dfc + matrix_size, *a = y_dfc + matrix_size, *b = a + matrix_size, *c = b + matrix_size, *d = c + matrix_size;

		auto begin = chrono::steady_clock::now(), end = begin;
		for (int count = 0; count != 10000; ++count) {
			if (hipSuccess != (cudaStatus = iteration_step(&f_prev, &f_curr, &f_tmp, x_dfc, y_dfc, a, b, c, d, rx, ry, x_size, y_size))) {
				cerr << "On iteration " << count << " step kernell failed: " << endl;
				cerr << cudaStatus << " -- " << hipGetErrorString(cudaStatus) << endl;
				hipDeviceSynchronize();
				goto Clear;
			}
		}
		hipDeviceSynchronize();
		end = chrono::steady_clock::now();
		cerr << "Time consumed: " << chrono::duration <double, milli>(end - begin).count() << " ms" << endl;

		if (hipSuccess != (cudaStatus = hipMemcpy(f.data(), f_prev - matrix_shift, x_size*y_size * sizeof(float), hipMemcpyDeviceToHost))) {
			cout << "Can't copy data from f_prev to host:" << endl;
			cout << cudaStatus << " -- " << hipGetErrorString(cudaStatus) << endl;
			goto Clear;
		}
		else {
			ofstream ascii_out("./data/matrix.txt");
			ascii_out.precision(7); ascii_out.setf(ios::fixed, ios::floatfield);
			
				for (size_t x_idx = 0; x_idx != x_size; ++x_idx)
					for (size_t y_idx = 0; y_idx != y_size; ++y_idx)
					ascii_out << x_idx << " " << y_idx << " " << f[x_idx * y_size + y_idx] << endl;
		}
	}

Clear:
	if (NULL != gm_dev) hipFree(gm_dev);
	if (hipSuccess != (cudaStatus = hipDeviceReset())) {
		cerr << "Error in device process termination: " << endl;
		cerr << cudaStatus << " -- " << hipGetErrorString(cudaStatus) << endl;
	}

End:
	return 0;
}