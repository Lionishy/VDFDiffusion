#include "hip/hip_runtime.h"

#include "StridedThomsonSolver.cuh"

#include <hip/hip_runtime.h>

template <typename T>
__device__ void strided_set_initial_matrix(T *a, T *b, T *c, T *d, size_t size, size_t stride) {
	a[0] = T(0); b[0] = T(3); c[0] = T(1); d[0] = T(4);
	for (size_t idx = 1; idx != size - 1; ++idx) {
		size_t stride_idx = stride * idx;
		a[stride_idx] = T(1);
		b[stride_idx] = T(2);
		c[stride_idx] = T(1);
		d[stride_idx] = T(4);
	}

	size_t last_idx = stride * (size - 1);
	a[last_idx] = T(1);
	b[last_idx] = T(3);
	c[last_idx] = T(0);
	d[last_idx] = T(4);
}

template <typename T>
__global__ void strided_accurate_thomson_sweep_test_kernell(T *mem, size_t size, size_t span, size_t loop_count) {
	size_t shift = threadIdx.x + blockDim.x * blockIdx.x;
	size_t grid_size = size * span, stride = span;
	T *f = mem, *dfc = f + grid_size, *a = dfc + grid_size, *b = a + grid_size, *c = b + grid_size, *d = c + grid_size;
	strided_set_initial_matrix(a + shift, b + shift, c + shift, d + shift, size, stride);
	iki::math::device::accurate_strided_thomson_sweep(a + shift, b + shift, c + shift, d + shift, f + shift, size, stride);


	/*size_t shift = threadIdx.x + blockDim.x * blockIdx.x;
	size_t grid_size = size * span, stride = span;
	T *f = mem, *dfc = f + grid_size, *a = dfc + grid_size, *b = a + grid_size, *c = b + grid_size, *d = c + grid_size;
	strided_set_initial_state(f + shift, dfc + shift, size, stride);
	for (; loop_count != 0; --loop_count)
		iki::diffusion::device::strided_diffusion_step(f + shift, dfc + shift, a + shift, b + shift, c + shift, d + shift, T(1.), size, stride);*/
}

#include <iostream>
#include <vector>
#include <algorithm>
#include <vector>
#include <fstream>
#include <sstream>
#include <chrono>

int main() {
	using namespace std;

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		cout << "Error in starting cuda device" << endl;
		goto End;
	}

	{
		size_t size = 1024, span = 1024;

		//we need to allocate a number of grids size*span elements each
		//f_curr_dev, d_curr_dev
		//we also need to allocate 4 grids for the Thomson sweep method
		//a b c and d
		//6 grids in total
		float *mem_dev = NULL; //a pointer to the device global memory 
		if (hipSuccess != hipMalloc((void **)&mem_dev, 6 * size * span * sizeof(float))) {
			cout << "Can't allocate memory for function: " << 6 * size * span * sizeof(float) / 1024 << " Kb" << endl;
			goto Clear;
		}

		{
			unsigned threads_count = 512, blocks_count = span / threads_count;
			auto begin = chrono::steady_clock::now(), end = begin;
			strided_accurate_thomson_sweep_test_kernell <<<1, 1>>> (mem_dev, size, span, 1u);
			if (hipSuccess != (cudaStatus = hipGetLastError())) {
				cout << "Kernell launch failed: " << cudaStatus << " -- " << hipGetErrorString(cudaStatus) << endl;
				goto Clear;
			}
			hipDeviceSynchronize();
			end = chrono::steady_clock::now();
			cout << "Time consumed: " << chrono::duration <double, milli>(end - begin).count() << " ms" << endl;
		}
		if (hipSuccess != (cudaStatus = hipGetLastError())) {
			cout << "Kernell execution failed: " << cudaStatus << " -- " << hipGetErrorString(cudaStatus) << endl;
			goto Clear;
		}
		else {
			cout << "Calculation Success!" << endl;
			vector<float> f_next(size * span); //to temporary save data from GPU for the further export into .txt file
			ofstream ascii_out("./data/f.txt"); ascii_out.precision(7); ascii_out.setf(std::ios::fixed, std::ios::floatfield);
			if (hipSuccess != hipMemcpy(f_next.data(), mem_dev, span * size * sizeof(float), hipMemcpyDeviceToHost)) {
				cout << "Memory copy device->host failed!" << endl;
			}
			else {
				for (size_t row_idx = 0; row_idx != span; ++row_idx)
					for (size_t idx = 0; idx != size; ++idx)
						ascii_out << row_idx << ' ' << idx << ' ' << f_next[row_idx + idx*span] << '\n';
				ascii_out << endl;
			}
		}


	Clear:;
		if (mem_dev != NULL) hipFree(mem_dev);
	}

	if (hipSuccess != hipDeviceReset()) {
		cout << "Error in device process termination!" << endl;
	}
End:;
	return 0;
}